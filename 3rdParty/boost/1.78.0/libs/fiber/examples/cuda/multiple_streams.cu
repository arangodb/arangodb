#include "hip/hip_runtime.h"

//          Copyright Oliver Kowalke 2013.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE_1_0.txt or copy at
//          http://www.boost.org/LICENSE_1_0.txt)


#include <chrono>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <random>
#include <tuple>

#include <hip/hip_runtime.h>

#include <boost/assert.hpp>
#include <boost/bind.hpp>
#include <boost/intrusive_ptr.hpp>

#include <boost/fiber/all.hpp>
#include <boost/fiber/cuda/waitfor.hpp>

__global__
void vector_add( int * a, int * b, int * c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    try {
        bool done = false;
        boost::fibers::fiber f1( [&done]{
                std::cout << "f1: entered" << std::endl;
                try {
                    hipStream_t stream0, stream1;
                    hipStreamCreate( & stream0);
                    hipStreamCreate( & stream1);
                    int size = 1024 * 1024;
                    int full_size = 20 * size;
                    int * host_a, * host_b, * host_c;
                    hipHostAlloc( & host_a, full_size * sizeof( int), hipHostMallocDefault);
                    hipHostAlloc( & host_b, full_size * sizeof( int), hipHostMallocDefault);
                    hipHostAlloc( & host_c, full_size * sizeof( int), hipHostMallocDefault);
                    int * dev_a0, * dev_b0, * dev_c0;
                    int * dev_a1, * dev_b1, * dev_c1;
                    hipMalloc( & dev_a0, size * sizeof( int) );
                    hipMalloc( & dev_b0, size * sizeof( int) );
                    hipMalloc( & dev_c0, size * sizeof( int) );
                    hipMalloc( & dev_a1, size * sizeof( int) );
                    hipMalloc( & dev_b1, size * sizeof( int) );
                    hipMalloc( & dev_c1, size * sizeof( int) );
                    std::minstd_rand generator;
                    std::uniform_int_distribution<> distribution(1, 6);
                    for ( int i = 0; i < full_size; ++i) {
                        host_a[i] = distribution( generator);
                        host_b[i] = distribution( generator);
                    }
                    for ( int i = 0; i < full_size; i += 2 * size) {
                        hipMemcpyAsync( dev_a0, host_a + i, size * sizeof( int), hipMemcpyHostToDevice, stream0);
                        hipMemcpyAsync( dev_a1, host_a + i + size, size * sizeof( int), hipMemcpyHostToDevice, stream1);
                        hipMemcpyAsync( dev_b0, host_b + i, size * sizeof( int), hipMemcpyHostToDevice, stream0);
                        hipMemcpyAsync( dev_b1, host_b + i + size, size * sizeof( int), hipMemcpyHostToDevice, stream1);
                        vector_add<<< size / 256, 256, 0, stream0 >>>( dev_a0, dev_b0, dev_c0, size);
                        vector_add<<< size / 256, 256, 0, stream1 >>>( dev_a1, dev_b1, dev_c1, size);
                        hipMemcpyAsync( host_c + i, dev_c0, size * sizeof( int), hipMemcpyDeviceToHost, stream0);
                        hipMemcpyAsync( host_c + i + size, dev_c1, size * sizeof( int), hipMemcpyDeviceToHost, stream1);
                    }
                    auto results = boost::fibers::cuda::waitfor_all( stream0, stream1);
                    for ( auto & result : results) {
                        BOOST_ASSERT( stream0 == std::get< 0 >( result) || stream1 == std::get< 0 >( result) );
                        BOOST_ASSERT( hipSuccess == std::get< 1 >( result) );
                    }
                    std::cout << "f1: GPU computation finished" << std::endl;
                    hipHostFree( host_a);
                    hipHostFree( host_b);
                    hipHostFree( host_c);
                    hipFree( dev_a0);
                    hipFree( dev_b0);
                    hipFree( dev_c0);
                    hipFree( dev_a1);
                    hipFree( dev_b1);
                    hipFree( dev_c1);
                    hipStreamDestroy( stream0);
                    hipStreamDestroy( stream1);
                    done = true;
                } catch ( std::exception const& ex) {
                    std::cerr << "exception: " << ex.what() << std::endl;
                }
                std::cout << "f1: leaving" << std::endl;
        });
        boost::fibers::fiber f2([&done]{
            std::cout << "f2: entered" << std::endl;
            while ( ! done) {
                std::cout << "f2: sleeping" << std::endl;
                boost::this_fiber::sleep_for( std::chrono::milliseconds( 1 ) );
            }
            std::cout << "f2: leaving" << std::endl;
        });
        f1.join();
        f2.join();
        std::cout << "done." << std::endl;
        return EXIT_SUCCESS;
    } catch ( std::exception const& e) {
        std::cerr << "exception: " << e.what() << std::endl;
    } catch (...) {
        std::cerr << "unhandled exception" << std::endl;
    }
	return EXIT_FAILURE;
}
