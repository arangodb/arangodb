//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include "perf.hpp"

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> h_vec = generate_random_vector<int>(PERF_N);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        d_vec = h_vec;

        t.start();
        thrust::inclusive_scan(d_vec.begin(), d_vec.end(), d_vec.begin());
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    return 0;
}
