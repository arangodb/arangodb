//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/unique.h>

#include "perf.hpp"

int rand_int()
{
    return static_cast<int>((rand() / double(RAND_MAX)) * 25.0);
}

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> h_vec(PERF_N);
    std::generate(h_vec.begin(), h_vec.end(), rand_int);

    thrust::device_vector<int> d_vec(PERF_N);

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        d_vec = h_vec;

        t.start();
        thrust::unique(d_vec.begin(), d_vec.end());
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;

    return 0;
}
