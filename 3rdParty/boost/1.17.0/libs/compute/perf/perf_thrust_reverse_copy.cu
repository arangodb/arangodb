//---------------------------------------------------------------------------//
// Copyright (c) 2015 Jakub Szuppe <j.szuppe@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/reverse.h>

#include "perf.hpp"

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> h_vec = generate_random_vector<int>(PERF_N);

	// transfer data to the device
    thrust::device_vector<int> d_vec;   
    d_vec = h_vec;
    
    // device vector for reversed data
    thrust::device_vector<int> d_reversed_vec(PERF_N);
    
    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        thrust::reverse_copy(d_vec.begin(), d_vec.end(), d_reversed_vec.begin());
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;

    return 0;
}
