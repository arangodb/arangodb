#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <iostream>
#include <iterator>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>

#include "perf.hpp"

int rand_int()
{
    return static_cast<int>((rand() / double(RAND_MAX)) * 25.0);
}

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> v1(std::floor(PERF_N / 2.0));
    thrust::host_vector<int> v2(std::ceil(PERF_N / 2.0));
    std::generate(v1.begin(), v1.end(), rand_int);
    std::generate(v2.begin(), v2.end(), rand_int);
    std::sort(v1.begin(), v1.end());
    std::sort(v2.begin(), v2.end());

    // transfer data to the device
    thrust::device_vector<int> gpu_v1 = v1;
    thrust::device_vector<int> gpu_v2 = v2;
    thrust::device_vector<int> gpu_v3(PERF_N);

    thrust::device_vector<int>::iterator gpu_v3_end;

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        gpu_v3_end = thrust::set_difference(
            gpu_v1.begin(), gpu_v1.end(),
            gpu_v2.begin(), gpu_v2.end(),
            gpu_v3.begin()
        );
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;
    std::cout << "size: " << thrust::distance(gpu_v3.begin(), gpu_v3_end) << std::endl;

    return 0;
}
