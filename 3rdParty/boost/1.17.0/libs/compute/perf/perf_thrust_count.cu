//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <iostream>
#include <vector>

#include <thrust/count.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "perf.hpp"

int rand_int()
{
    return static_cast<int>((rand() / double(RAND_MAX)) * 25.0);
}

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);
    std::cout << "size: " << PERF_N << std::endl;

    // create vector of random numbers on the host
    thrust::host_vector<int> host_vector(PERF_N);
    thrust::generate(host_vector.begin(), host_vector.end(), rand_int);

    thrust::device_vector<int> v = host_vector;

    size_t count = 0;
    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        count = thrust::count(v.begin(), v.end(), 4);
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;
    std::cout << "count: " << count << std::endl;

    return 0;
}
