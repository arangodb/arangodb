//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <iostream>
#include <iterator>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>

#include "perf.hpp"

struct saxpy_functor : public thrust::binary_function<float,float,float>
{
    const float a;

    saxpy_functor(float _a) : a(_a) {}

    __host__ __device__
    float operator()(const float& x, const float& y) const
    {
        return a * x + y;
    }
};

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> host_x(PERF_N);
    thrust::host_vector<int> host_y(PERF_N);
    std::generate(host_x.begin(), host_x.end(), rand);
    std::generate(host_y.begin(), host_y.end(), rand);

    // transfer data to the device
    thrust::device_vector<int> device_x = host_x;
    thrust::device_vector<int> device_y = host_y;

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        thrust::transform(device_x.begin(), device_x.end(), device_y.begin(), device_y.begin(), saxpy_functor(2.5f));
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;

    // transfer data back to host
    thrust::copy(device_x.begin(), device_x.end(), host_x.begin());
    thrust::copy(device_y.begin(), device_y.end(), host_y.begin());

    return 0;
}
