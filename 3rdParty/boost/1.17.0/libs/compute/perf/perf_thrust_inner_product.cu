//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <iostream>
#include <iterator>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>

#include "perf.hpp"

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> host_x(PERF_N);
    thrust::host_vector<int> host_y(PERF_N);
    std::generate(host_x.begin(), host_x.end(), rand);
    std::generate(host_y.begin(), host_y.end(), rand);

    // transfer data to the device
    thrust::device_vector<int> device_x = host_x;
    thrust::device_vector<int> device_y = host_y;

    int product = 0;
    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        product = thrust::inner_product(
            device_x.begin(), device_x.end(), device_y.begin(), 0
        );
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;
    std::cout << "product: " << product << std::endl;

    return 0;
}
