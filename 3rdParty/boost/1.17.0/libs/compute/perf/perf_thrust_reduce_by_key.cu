//---------------------------------------------------------------------------//
// Copyright (c) 2015 Jakub Szuppe <j.szuppe@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>

#include "perf.hpp"

int rand_int()
{
    return static_cast<int>((rand() / double(RAND_MAX)) * 25.0);
}

struct unique_key {
  int current;
  int avgValuesNoPerKey;

  unique_key()
  {
      current = 0;
      avgValuesNoPerKey = 512;
  }

  int operator()()
  {
      double p = double(1.0) / static_cast<double>(avgValuesNoPerKey);
      if((rand() / double(RAND_MAX)) <= p)
          return ++current;
      return current;
  }
} UniqueKey;

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    
    // create vector of keys and random values
    thrust::host_vector<int> host_keys(PERF_N);
    thrust::host_vector<int> host_values(PERF_N);
    std::generate(host_keys.begin(), host_keys.end(), UniqueKey);
    std::generate(host_values.begin(), host_values.end(), rand_int);
    
    // transfer data to the device
    thrust::device_vector<int> device_keys = host_keys;
    thrust::device_vector<int> device_values = host_values;

    // create device vectors for the results
    thrust::device_vector<int> device_keys_results(PERF_N);
    thrust::device_vector<int> device_values_results(PERF_N);

    typedef typename thrust::device_vector<int>::iterator iterType;
    thrust::pair<iterType, iterType> result;

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        result = thrust::reduce_by_key(device_keys.begin(),
                                       device_keys.end(),
                                       device_values.begin(),
                                       device_keys_results.begin(),
                                       device_values_results.begin());
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;
    
    size_t result_size = thrust::distance(device_keys_results.begin(), result.first);
    if(result_size != static_cast<size_t>(host_keys[PERF_N-1] + 1)){
        std::cout << "ERROR: "
                  << "wrong number of keys"
                  << std::endl;
        return -1;
    }

    return 0;
}
