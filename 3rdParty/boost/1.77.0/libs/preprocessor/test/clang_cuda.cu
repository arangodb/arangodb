# /* **************************************************************************
#  *                                                                          *
#  *     (C) Copyright Edward Diener 2019.
#  *     Distributed under the Boost Software License, Version 1.0. (See
#  *     accompanying file LICENSE_1_0.txt or copy at
#  *     http://www.boost.org/LICENSE_1_0.txt)
#  *                                                                          *
#  ************************************************************************** */
#
# /* See http://www.boost.org for most recent version. */
#
# include <libs/preprocessor/test/test_macro.h>

#if defined(__clang__) && defined(__HIPCC__) && defined(__CUDA__)

BEGIN BOOST_PP_VARIADICS == 1 END

#else

BEGIN 0 == 1 END

#endif

int main(void) {
    return 0;
}
