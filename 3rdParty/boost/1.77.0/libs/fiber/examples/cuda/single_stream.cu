#include "hip/hip_runtime.h"

//          Copyright Oliver Kowalke 2017.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE_1_0.txt or copy at
//          http://www.boost.org/LICENSE_1_0.txt)

#include <chrono>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <random>
#include <tuple>

#include <hip/hip_runtime.h>

#include <boost/assert.hpp>
#include <boost/bind.hpp>
#include <boost/intrusive_ptr.hpp>

#include <boost/fiber/all.hpp>
#include <boost/fiber/cuda/waitfor.hpp>

__global__
void vector_add( int * a, int * b, int * c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    try {
        bool done = false;
        boost::fibers::fiber f1([&done]{
            std::cout << "f1: entered" << std::endl;
            try {
                hipStream_t stream;
                hipStreamCreate( & stream);
                int size = 1024 * 1024;
                int full_size = 20 * size;
                int * host_a, * host_b, * host_c;
                hipHostAlloc( & host_a, full_size * sizeof( int), hipHostMallocDefault);
                hipHostAlloc( & host_b, full_size * sizeof( int), hipHostMallocDefault);
                hipHostAlloc( & host_c, full_size * sizeof( int), hipHostMallocDefault);
                int * dev_a, * dev_b, * dev_c;
                hipMalloc( & dev_a, size * sizeof( int) );
                hipMalloc( & dev_b, size * sizeof( int) );
                hipMalloc( & dev_c, size * sizeof( int) );
                std::minstd_rand generator;
                std::uniform_int_distribution<> distribution(1, 6);
                for ( int i = 0; i < full_size; ++i) {
                    host_a[i] = distribution( generator);
                    host_b[i] = distribution( generator);
                }
                for ( int i = 0; i < full_size; i += size) {
                    hipMemcpyAsync( dev_a, host_a + i, size * sizeof( int), hipMemcpyHostToDevice, stream);
                    hipMemcpyAsync( dev_b, host_b + i, size * sizeof( int), hipMemcpyHostToDevice, stream);
                    vector_add<<< size / 256, 256, 0, stream >>>( dev_a, dev_b, dev_c, size);
                    hipMemcpyAsync( host_c + i, dev_c, size * sizeof( int), hipMemcpyDeviceToHost, stream);
                }
                auto result = boost::fibers::cuda::waitfor_all( stream);
                BOOST_ASSERT( stream == std::get< 0 >( result) );
                BOOST_ASSERT( hipSuccess == std::get< 1 >( result) );
                std::cout << "f1: GPU computation finished" << std::endl;
                hipHostFree( host_a);
                hipHostFree( host_b);
                hipHostFree( host_c);
                hipFree( dev_a);
                hipFree( dev_b);
                hipFree( dev_c);
                hipStreamDestroy( stream);
                done = true;
            } catch ( std::exception const& ex) {
                std::cerr << "exception: " << ex.what() << std::endl;
            }
            std::cout << "f1: leaving" << std::endl;
        });
        boost::fibers::fiber f2([&done]{
            std::cout << "f2: entered" << std::endl;
            while ( ! done) {
                std::cout << "f2: sleeping" << std::endl;
                boost::this_fiber::sleep_for( std::chrono::milliseconds( 1 ) );
            }
            std::cout << "f2: leaving" << std::endl;
        });
        f1.join();
        f2.join();
        std::cout << "done." << std::endl;
        return EXIT_SUCCESS;
    } catch ( std::exception const& e) {
        std::cerr << "exception: " << e.what() << std::endl;
    } catch (...) {
        std::cerr << "unhandled exception" << std::endl;
    }
	return EXIT_FAILURE;
}
