#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <iostream>
#include <iterator>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/merge.h>
#include <thrust/sort.h>

#include "perf.hpp"

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> v1(std::floor(PERF_N / 2.0));
    thrust::host_vector<int> v2(std::ceil(PERF_N / 2.0));
    std::generate(v1.begin(), v1.end(), rand);
    std::generate(v2.begin(), v2.end(), rand);
    std::sort(v1.begin(), v1.end());
    std::sort(v2.begin(), v2.end());

    // transfer data to the device
    thrust::device_vector<int> gpu_v1 = v1;
    thrust::device_vector<int> gpu_v2 = v2;
    thrust::device_vector<int> gpu_v3(PERF_N);

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        thrust::merge(
            gpu_v1.begin(), gpu_v1.end(),
            gpu_v2.begin(), gpu_v2.end(),
            gpu_v3.begin()
        );
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;

    thrust::host_vector<int> check_v3 = gpu_v3;

    thrust::host_vector<int> v3(PERF_N);
    std::merge(v1.begin(), v1.end(), v2.begin(), v2.end(), v3.begin());
    bool ok = std::equal(check_v3.begin(), check_v3.end(), v3.begin());
    if(!ok){
        std::cerr << "ERROR: merged ranges different" << std::endl;
        return -1;
    }

    return 0;
}
