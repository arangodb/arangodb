//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "perf.hpp"

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> h_vec = generate_random_vector<int>(PERF_N);

    // transfer data to the device
    thrust::device_vector<int> d_vec;

    size_t rotate_distance = PERF_N / 2;

    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        d_vec = h_vec;

        t.start();
        // there is no thrust::rotate() so we implement it manually with copy()
        thrust::device_vector<int> tmp(d_vec.begin(), d_vec.begin() + rotate_distance);
        thrust::copy(d_vec.begin() + rotate_distance, d_vec.end(), d_vec.begin());
        thrust::copy(tmp.begin(), tmp.end(), d_vec.begin() + rotate_distance);
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    return 0;
}
