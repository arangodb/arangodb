//---------------------------------------------------------------------------//
// Copyright (c) 2013-2014 Kyle Lutz <kyle.r.lutz@gmail.com>
//
// Distributed under the Boost Software License, Version 1.0
// See accompanying file LICENSE_1_0.txt or copy at
// http://www.boost.org/LICENSE_1_0.txt
//
// See http://boostorg.github.com/compute for more information.
//---------------------------------------------------------------------------//

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>

#include "perf.hpp"

int main(int argc, char *argv[])
{
    perf_parse_args(argc, argv);

    std::cout << "size: " << PERF_N << std::endl;
    thrust::host_vector<int> h_vec = generate_random_vector<int>(PERF_N);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    int sum = 0;
    perf_timer t;
    for(size_t trial = 0; trial < PERF_TRIALS; trial++){
        t.start();
        sum = thrust::reduce(d_vec.begin(), d_vec.end());
        hipDeviceSynchronize();
        t.stop();
    }
    std::cout << "time: " << t.min_time() / 1e6 << " ms" << std::endl;
    std::cout << "sum: " << sum << std::endl;

    return 0;
}
